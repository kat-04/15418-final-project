#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <string>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>
#include <iostream>
#include <cmath>
#include <bitset>
#include "cube.h"
#include "fileLoader.h"
#include "golCuda.h"

#define BLOCK_SIZE 256
// Comment for faster runtimes
//#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#else
#define cudaCheckError(ans) ans
#endif

struct GlobalConstants {
    uint64_t sideLength;
    bool isMoore;
    int numStates;

    bool* ruleset;
    uint32_t* minMaxs;

    uint8_t* inputData;
    uint8_t* outputData;
};

__constant__ GlobalConstants cuConstIterationParams;

// do an iteration of the algorithm with Moore neighborhoods 
// each kernel call does one int's voxels (so 8 as each bit stores a state)
__global__ void kernelDoIterationMoore(bool doBoundingBox) {
    uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t n = cuConstIterationParams.sideLength;
    uint64_t boundedX;
    uint64_t boundedY;
    uint64_t boundedZ;
    if (index >= (n * n * n + 7) / 8) {
        return;
    }

    // get bounding box size
    if (doBoundingBox) {
        boundedX = 1 + cuConstIterationParams.minMaxs[3] - cuConstIterationParams.minMaxs[0];
        boundedY = 1 + cuConstIterationParams.minMaxs[4] - cuConstIterationParams.minMaxs[1];
        boundedZ = 1 + cuConstIterationParams.minMaxs[5] - cuConstIterationParams.minMaxs[2]; 
        if (index >= (boundedX * boundedY * boundedZ + 7) / 8) {
            return;
        }   
    }    

    // index of the first thing in the bit array
    uint64_t bitIndex = 8 * index;
    uint64_t neighborBitIndex;
    uint64_t neighborLinIndex;
    int neighborBit;
    uint8_t mask = 1;
    int numAlive = 0;
    int status;
    
    // for each bit in the current int
    for (int bit = 0; bit < 8; bit++) {
        if (doBoundingBox && bitIndex + bit >= boundedX * boundedY * boundedZ) {
            break;
        }
        uint64_t x;
        uint64_t y;
        uint64_t z;
        if (doBoundingBox) {
            // get shifted bounds based on global min/max
            x = (((bitIndex + bit) % boundedX) + cuConstIterationParams.minMaxs[0]);
            y = (((bitIndex + bit ) / boundedX) % boundedY) + cuConstIterationParams.minMaxs[1];
            z = (((bitIndex + bit ) / (boundedX * boundedY)) % boundedZ) + cuConstIterationParams.minMaxs[2];
        } else {
            x = (((bitIndex + bit) % n));
            y = (((bitIndex + bit ) / n) % n);
            z = (((bitIndex + bit ) / (n * n)) % n);
        }
        
        for (uint64_t i = (x == 0) ? 0 : x - 1; i <= x + 1; i++) {
            for (uint64_t j = (y == 0) ? 0 : y - 1; j <= y + 1; j++) {
                for (uint64_t k = (z == 0) ? 0 : z - 1; k <= z + 1; k++) {
                    if (i < n && j < n && k < n) {
                        if (!(x == i && y == j && z == k)) { //don't include self
                            neighborBitIndex = (k * n * n) + (j * n) + i;
                            neighborLinIndex = neighborBitIndex / 8;
                            neighborBit = neighborBitIndex % 8;

                            numAlive += (cuConstIterationParams.inputData[neighborLinIndex] >> (7 - neighborBit)) & mask;
                        }
                    }
                }
            }
        }
        uint64_t shiftedLinIndex = (z * n * n + y * n + x) / 8;
        uint64_t shiftedBit = (z * n * n + y * n + x) % 8;
        if ((cuConstIterationParams.inputData[shiftedLinIndex] >> (7 - shiftedBit)) & mask) {
            // voxel was previously alive
            status = cuConstIterationParams.ruleset[27 + numAlive] ? 1 : 0;
            if (status) {
                // stays alive
                cuConstIterationParams.outputData[shiftedLinIndex] = cuConstIterationParams.outputData[shiftedLinIndex] | (status << (7 - shiftedBit));
            } else {
                // now dead
                cuConstIterationParams.outputData[shiftedLinIndex] = cuConstIterationParams.outputData[shiftedLinIndex] & ~(1 << (7 - shiftedBit));
            }
        } else {
            // voxel was previously dead
            status = cuConstIterationParams.ruleset[numAlive] ? 1 : 0;
            cuConstIterationParams.outputData[shiftedLinIndex] = cuConstIterationParams.outputData[shiftedLinIndex] | (status << (7 - shiftedBit));
        }
        numAlive = 0;
    }
}

// do an iteration of the algorithm with Von Neumann neighborhoods 
// each kernel call does one int's voxels (so 8 as each bit stores a state)
__global__ void kernelDoIterationVonNeumann(bool doBoundingBox) {
    uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t n = cuConstIterationParams.sideLength;
    if (index >= (n * n * n + 7) / 8) {
        return;
    }

    uint64_t boundedX;
    uint64_t boundedY;
    uint64_t boundedZ;

    // get bounding box size
    if (doBoundingBox) {
        boundedX = 1 + cuConstIterationParams.minMaxs[3] - cuConstIterationParams.minMaxs[0];
        boundedY = 1 + cuConstIterationParams.minMaxs[4] - cuConstIterationParams.minMaxs[1];
        boundedZ = 1 + cuConstIterationParams.minMaxs[5] - cuConstIterationParams.minMaxs[2]; 
        if (index >= (boundedX * boundedY * boundedZ + 7) / 8) {
            return;
        }   
    }  
    
    uint64_t bitIndex = index * 8;
    uint64_t neighborBitIndex;
    uint64_t neighborLinIndex;
    int neighborBit;
    uint8_t mask = 1;
    int numAlive = 0;
    int status;

    for (int bit = 0; bit < 8; bit++) {
        if (doBoundingBox && bitIndex + bit >= boundedX * boundedY * boundedZ) {
            break;
        }
        uint64_t x;
        uint64_t y;
        uint64_t z;

        if (doBoundingBox) {
            // get shifted bounds based on global min/max
            x = (((bitIndex + bit) % boundedX) + cuConstIterationParams.minMaxs[0]);
            y = (((bitIndex + bit ) / boundedX) % boundedY) + cuConstIterationParams.minMaxs[1];
            z = (((bitIndex + bit ) / (boundedX * boundedY)) % boundedZ) + cuConstIterationParams.minMaxs[2];
        } else {
            x = (((bitIndex + bit) % n));
            y = (((bitIndex + bit ) / n) % n);
            z = (((bitIndex + bit ) / (n * n)) % n);
        }

        neighborBitIndex = (z * n * n) + (y * n) + x - 1;
        neighborLinIndex = neighborBitIndex / 8;
        neighborBit = neighborBitIndex % 8;
        if (x > 0 && x < n) numAlive += (cuConstIterationParams.inputData[neighborLinIndex] >> (7 - neighborBit)) & mask;
        
        neighborBitIndex = (z * n * n) + (y * n) + x + 1;
        neighborLinIndex = neighborBitIndex / 8;
        neighborBit = neighborBitIndex % 8;
        if (x + 1 < n) numAlive += (cuConstIterationParams.inputData[neighborLinIndex] >> (7 - neighborBit)) & mask;

        neighborBitIndex = (z * n * n) + ((y - 1) * n) + x;
        neighborLinIndex = neighborBitIndex / 8;
        neighborBit = neighborBitIndex % 8;
        if (y > 0 && y < n) numAlive += (cuConstIterationParams.inputData[neighborLinIndex] >> (7 - neighborBit)) & mask;

        neighborBitIndex = (z * n * n) + ((y + 1) * n) + x;
        neighborLinIndex = neighborBitIndex / 8;
        neighborBit = neighborBitIndex % 8;
        if (y + 1 < n) numAlive += (cuConstIterationParams.inputData[neighborLinIndex] >> (7 - neighborBit)) & mask;

        neighborBitIndex = ((z - 1) * n * n) + (y * n) + x;
        neighborLinIndex = neighborBitIndex / 8;
        neighborBit = neighborBitIndex % 8;
        if (z > 0 && z < n) numAlive += (cuConstIterationParams.inputData[neighborLinIndex] >> (7 - neighborBit)) & mask;

        neighborBitIndex = ((z + 1) * n * n) + (y * n) + x;
        neighborLinIndex = neighborBitIndex / 8;
        neighborBit = neighborBitIndex % 8;
        if (z + 1 < n) numAlive += (cuConstIterationParams.inputData[neighborLinIndex] >> (7 - neighborBit)) & mask;

        uint64_t shiftedLinIndex = (z * n * n + y * n + x) / 8;
        uint64_t shiftedBit = (z * n * n + y * n + x) % 8;
        if ((cuConstIterationParams.inputData[shiftedLinIndex] >> (7 - shiftedBit)) & mask) {
            status = cuConstIterationParams.ruleset[27 + numAlive] ? 1 : 0;
            if (status) {
                // alive
                cuConstIterationParams.outputData[shiftedLinIndex] = cuConstIterationParams.outputData[shiftedLinIndex] | (status << (7 - shiftedBit));
            } else {
                // dead
                cuConstIterationParams.outputData[shiftedLinIndex] = cuConstIterationParams.outputData[shiftedLinIndex] & ~(1 << (7 - shiftedBit));
            }
        } else {
            status = cuConstIterationParams.ruleset[numAlive] ? 1 : 0;
            cuConstIterationParams.outputData[shiftedLinIndex] = cuConstIterationParams.outputData[shiftedLinIndex] | (status << (7 - shiftedBit));
        }

        numAlive = 0;
    }
}

// get the new global bounds of the alive voxels
__global__ void kernelGetGlobalBounds() {
    uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t n = cuConstIterationParams.sideLength;
    if (index >= n) {
        return;
    }
    
    // first get local min/max values for this kernel call
    uint64_t localMinX = n - 1;
    uint64_t localMinY = n - 1;
    uint64_t localMinZ = n - 1;
    uint64_t localMaxX = 0;
    uint64_t localMaxY = 0;
    uint64_t localMaxZ = 0;

    uint64_t bit_index;
    // Bit and index in array of neighbor
    uint64_t linIndex;
    uint8_t bit;
    for (uint64_t y = 0; y < n; y++) {
        for (uint64_t z = 0; z < n; z++) {
            if (y >= n || z >= n) {
                continue;
            }

            bit_index = index + y * n + z * n * n;
            linIndex = bit_index / 8;
            bit = bit_index % 8;

            uint8_t mask = 1;
            uint8_t alive = ((cuConstIterationParams.inputData[linIndex] >> (7 - bit))) & mask;
            if (alive) {
                // update local values
                localMinX = min(localMinX, (index == 0) ? (uint64_t)0 : index - 1);
                localMinY = min(localMinY, (y == 0) ? (uint64_t)0 : y - 1);
                localMinZ = min(localMinZ, (z == 0) ? (uint64_t)0 : z - 1);
                localMaxX = min(max(localMaxX, index + 1), n - 1);
                localMaxY = min(max(localMaxY, y + 1), n - 1);
                localMaxZ = min(max(localMaxZ, z + 1), n - 1);
            }
        }
    }
    
    // atomically update global values with best local values
    atomicMin(&(cuConstIterationParams.minMaxs[0]), (uint32_t)localMinX);
    atomicMin(&(cuConstIterationParams.minMaxs[1]), (uint32_t)localMinY);
    atomicMin(&(cuConstIterationParams.minMaxs[2]), (uint32_t)localMinZ);
    atomicMax(&(cuConstIterationParams.minMaxs[3]), (uint32_t)localMaxX);
    atomicMax(&(cuConstIterationParams.minMaxs[4]), (uint32_t)localMaxY);
    atomicMax(&(cuConstIterationParams.minMaxs[5]), (uint32_t)localMaxZ);    
}



GolCuda::GolCuda() {
    sideLength = 0;
    isMoore = true;
    numStates = 0;

    cube = NULL;
    ruleset = NULL;
    inputData = NULL;
    minMaxs = NULL;

    cudaDeviceMinMaxs = NULL;
    cudaDeviceInputData = NULL;
    cudaDeviceOutputData = NULL;
    cudaDeviceRuleset = NULL;
}

GolCuda::~GolCuda() {

    if (cube) {
        delete [] cube;
    }
    if (ruleset) {
        delete [] ruleset;
    } 
    if (inputData) {
        delete [] inputData;
    }
    if (minMaxs) {
        delete [] minMaxs;
    }
    if (cudaDeviceInputData) {
        cudaCheckError(hipFree(cudaDeviceRuleset));
        cudaCheckError(hipFree(cudaDeviceInputData));
        cudaCheckError(hipFree(cudaDeviceOutputData));
        cudaCheckError(hipFree(cudaDeviceMinMaxs));
    }
}

void
GolCuda::clearOutputCube() {
    cube->clear();
}

void
GolCuda::allocOutputCube(uint64_t sideLength) {
    printf("Size of data: %f MB\n", sizeof(uint8_t) * (((sideLength * sideLength * sideLength + 7) / 8) / (1024.f * 1024.f)));
    if (cube) delete cube;
    cube = new Cube(sideLength);
    if (!cube) {
        std::cerr << "Cube allocation failed" << std::endl;
    }
}
 
Cube*
GolCuda::getCube() {

    // copy over result from device
    cudaCheckError(hipMemcpy(cube->data,
               cudaDeviceOutputData,
               sizeof(uint8_t) * ((sideLength * sideLength * sideLength + 7) / 8),
               hipMemcpyDeviceToHost));
    return cube;
}

int
GolCuda::loadInput(char* file, uint64_t n, char* outputDir) {
    return loadCubeInput(file, sideLength, ruleset, numStates, isMoore, inputData, n, outputDir, minMaxs);
}

void
GolCuda::setup() {

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("NVIDIA GeForce RTX 2080") == 0)
        {
            isFastGPU = true;
        }
    }
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }
    
    uint64_t cubeSize = (sideLength * sideLength * sideLength + 7) / 8;

    cudaCheckError(hipMalloc(&cudaDeviceRuleset, sizeof(bool) * 54));
    cudaCheckError(hipMalloc(&cudaDeviceInputData, sizeof(uint8_t) * cubeSize));
    cudaCheckError(hipMalloc(&cudaDeviceOutputData, sizeof(uint8_t) * cubeSize));
    cudaCheckError(hipMalloc(&cudaDeviceMinMaxs, sizeof(uint32_t) * 6));

    cudaCheckError(hipMemcpy(cudaDeviceRuleset, ruleset, sizeof(bool) * 54, hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(cudaDeviceInputData, inputData, sizeof(uint8_t) * cubeSize, hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(cudaDeviceMinMaxs, minMaxs, sizeof(uint32_t) * 6, hipMemcpyHostToDevice));
    

    // Initialize parameters in constant memory
    GlobalConstants params;
    params.sideLength = sideLength;
    params.isMoore = isMoore;
    params.numStates = numStates;
    params.minMaxs = cudaDeviceMinMaxs;
    params.inputData = cudaDeviceInputData;
    params.outputData = cudaDeviceOutputData;
    params.ruleset = cudaDeviceRuleset;
    
    cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(cuConstIterationParams), &params, sizeof(GlobalConstants)));
    
}
void GolCuda::updateBounds() {
    dim3 blockDim(sideLength);
    dim3 gridDim(1);

    // update minMaxs array to maxs with 0 and mins with n - 1
    minMaxs[0] = sideLength - 1;
    minMaxs[1] = sideLength - 1;
    minMaxs[2] = sideLength - 1;
    minMaxs[3] = 0;
    minMaxs[4] = 0;
    minMaxs[5] = 0;
    
    cudaCheckError(hipMemcpy(cudaDeviceMinMaxs, minMaxs, 6 * sizeof(uint32_t), hipMemcpyHostToDevice));
    
    kernelGetGlobalBounds<<<gridDim, blockDim>>>();
    
    cudaCheckError(hipMemcpy(minMaxs, cudaDeviceMinMaxs, 6 * sizeof(uint32_t), hipMemcpyDeviceToHost));
}

void 
GolCuda::advanceFrame() {
    // moves output to input for the next frame
    cudaCheckError(hipMemcpy(inputData,
        cudaDeviceOutputData,
        sizeof(uint8_t) * ((sideLength * sideLength * sideLength + 7) / 8),
        hipMemcpyDeviceToHost)); 

    cudaCheckError(hipMemcpy(cudaDeviceInputData, inputData, sizeof(uint8_t) * ((sideLength * sideLength * sideLength + 7) / 8), hipMemcpyHostToDevice));
}

void
GolCuda::doIteration(bool doBoundingBox) {    
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((((sideLength * sideLength * sideLength + 7) / 8) + blockDim.x - 1) / blockDim.x);

    if (doBoundingBox) {
        uint64_t inBoundsVolume = (1 + minMaxs[3] - minMaxs[0]) * (1 + minMaxs[4] - minMaxs[1]) * (1 + minMaxs[5] - minMaxs[2]);
        dim3 gridDim((((inBoundsVolume + 7) / 8) + blockDim.x - 1) / blockDim.x);
    }

    // do one iteration
    if (isMoore) {
        kernelDoIterationMoore<<<gridDim, blockDim>>>(doBoundingBox); 
    } else {
        kernelDoIterationVonNeumann<<<gridDim, blockDim>>>(doBoundingBox); 
    }  
}  