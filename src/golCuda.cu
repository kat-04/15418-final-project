//do stuff for global constants up top


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <string>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>
#include "cube.h"
#include "golCuda.h"

struct GlobalConstants {

    Cube* cubeData;

    int* sideLength;
    int* ruleset;
};

__constant__ GlobalConstants cuConstIterationParams;

GolCuda::GolCuda() {
    cubeData = NULL;
    sideLength = NULL;
    ruleset = NULL;

    cudaDeviceData = NULL;
    cudaDeviceSideLength = NULL;
    cudaDeviceRuleset = NULL;
}

GolCuda::~GolCuda() {

    if (cubeData) {
        delete cubeData;
    }
    if (ruleset) {
        delete [] ruleset;
    }
    if (cudaDeviceData) {
        hipFree(cudaDeviceData);
        hipFree(cudaDeviceSideLength);
        hipFree(cudaDeviceRuleset);
    }
}

void
GolCuda::clearResultCube() {

    // 256 threads per block is a healthy number
    // dim3 blockDim(16, 16, 1);
    // dim3 gridDim(
    //     (image->width + blockDim.x - 1) / blockDim.x,
    //     (image->height + blockDim.y - 1) / blockDim.y);

    // kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);

}

void
GolCuda::allocResultCube(int sideLength) {

    if (cubeData)
        delete cubeData;
    cubeData = new Cube(sideLength);
}

const Cube*
GolCuda::getResultCube() {

    // Need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    // printf("Copying image data from device\n");

    // hipMemcpy(cubeData,
    //            cudaDeviceData,
    //            //TODO,
    //            hipMemcpyDeviceToHost);

    // return cubeData;
    Cube* cube = new Cube(1);
    return cube;
}

void
GolCuda::loadInput(char* file) {
    // loadInput(file, sideLength, ruleset);
}

void
GolCuda::setup() {

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce RTX 2080") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&sideLength, sizeof(int));
    hipMalloc(&ruleset, sizeof(int) * 56);
    hipMalloc(&cudaDeviceSideLength, sizeof(int));
    hipMalloc(&cudaDeviceRuleset, sizeof(int) * 56);
    hipMalloc(&cudaDeviceData, sizeof(int) * (*sideLength) * (*sideLength) * (*sideLength));

    hipMemcpy(cudaDeviceSideLength, sideLength, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRuleset, ruleset, sizeof(int) * 56, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceData, cubeData, sizeof(int) * (*sideLength) * (*sideLength) * (*sideLength), hipMemcpyHostToDevice);
    

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.cubeData = cudaDeviceData;
    params.sideLength = cudaDeviceSideLength;
    params.ruleset = cudaDeviceRuleset;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstIterationParams), &params, sizeof(GlobalConstants));
}

void
GolCuda::doIteration() {
    return;
}